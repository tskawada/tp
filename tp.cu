#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudaarithm.hpp>
#include <iostream>
#include <stdio.h>

#include "include/config.hpp"
#include "include/imutil.hpp"
#include "include/eta.hpp"
#include "include/file.hpp"

using namespace std;
using namespace cv;
using namespace cv::cuda;

__global__ void blendAndCompareKernel(uchar* composite, uchar* image, uchar* nextImage, int cols, int rows, int threshold, float alpha) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int idx = y * cols + x;
        int baseIdx = idx * 3;

        uchar p_composite[3];
        uchar p_now[3];
        uchar p_next[3];

        p_composite[0] = composite[baseIdx];
        p_composite[1] = composite[baseIdx + 1];
        p_composite[2] = composite[baseIdx + 2];

        p_now[0] = image[baseIdx];
        p_now[1] = image[baseIdx + 1];
        p_now[2] = image[baseIdx + 2];

        p_next[0] = nextImage[baseIdx];
        p_next[1] = nextImage[baseIdx + 1];
        p_next[2] = nextImage[baseIdx + 2];

        float average_now = (p_now[0] + p_now[1] + p_now[2]) / 3.0f;
        float average_next = (p_next[0] + p_next[1] + p_next[2]) / 3.0f;

        if (average_next - average_now > threshold) {
            composite[baseIdx] = static_cast<uchar>(alpha * p_next[0] + (1.0f - alpha) * p_composite[0]);
            composite[baseIdx + 1] = static_cast<uchar>(alpha * p_next[1] + (1.0f - alpha) * p_composite[1]);
            composite[baseIdx + 2] = static_cast<uchar>(alpha * p_next[2] + (1.0f - alpha) * p_composite[2]);
        }
    }
}

int normal_timelapse(Config config, Input input_att) {
    ETA eta(config.num_of_frames);

    char image_name[100];    
    Mat image = Mat::zeros(config.height, config.width, CV_8UC3);

    // output movie file, non-compresse avi format
    VideoWriter writer;
    writer.open(
        config.output_file,
        VideoWriter::fourcc('I', 'Y', 'U', 'V'),
        config.fps,
        Size(config.width, config.height),
        true
    );
    if (!writer.isOpened()) return -1;

    for (int i = 1; i <= config.num_of_frames; i++) {
        eta.update();

        sprintf(image_name, "%s/%d.%s", config.input_folder.c_str(), i, input_att.file_extension.c_str());

        image = imread(image_name);
        writer.write(image);
    }
    writer.release();

    return 0;
}

int afterimage_timelapse(Config config, Input input_att) {
    ETA eta(config.num_of_frames);

    char image_name[100];
    GpuMat image;
    GpuMat compositeimage;

    VideoWriter writer;
    writer.open(
        config.output_file,
        VideoWriter::fourcc('I', 'Y', 'U', 'V'),
        config.fps,
        Size(config.width, config.height),
        true
    );
    if (!writer.isOpened()) return -1;

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((config.width + threadsPerBlock.x - 1) / threadsPerBlock.x, (config.height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    for (int i = 1; i <= config.num_of_frames; i++) {
        eta.update();
    
        sprintf(image_name, "%s/%d.%s", config.input_folder.c_str(), i, input_att.file_extension.c_str());
        Mat hostImage = imread(image_name);
    
        if (!hostImage.empty()) {
            GpuMat image;
            image.upload(hostImage);
    
            if (i == 1) {
                compositeimage = image.clone();
            } else {
                blendAndCompareKernel<<<numBlocks, threadsPerBlock>>>(compositeimage.data, image.data, compositeimage.data, config.width, config.height, config.threshold, config.alpha);
            }
    
            Mat compositeHost;
            compositeimage.download(compositeHost);
            writer.write(compositeHost);
        }
    }
    writer.release();

    return 0;
}

int main(int argc, char *argv[]) {
    Config config;
    if (argc == 1) config = config_loader("../assets/example.cfg");
    else if (argc == 2) config = config_loader(argv[1]);
    
    Input input_att = get_input_attributes(config.input_folder);
    if (config.num_of_frames == 0) config.num_of_frames = input_att.file_num;
    
    ImageSize image_size = get_image_size(config);
    config.width = image_size.width;
    config.height = image_size.height;

    config_printer(config);
    
    if (config.mode == 0) normal_timelapse(config, input_att);
    else if (config.mode == 1) afterimage_timelapse(config, input_att);

    return 0;
}
